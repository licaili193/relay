#include "hip/hip_runtime.h"
#include "image_processing.h"

namespace image_processing {

namespace {

__device__ float clamp(float x, float a, float b) {
  return max(a, min(b, x));
}

// Thread per block: 1D - 512
// Blocks per grid: 2D - height, (width + 511) / 512
__global__ void copyPixel3(size_t width, size_t height, uint8_t* d_img_1, uint8_t* d_img_2, uint8_t* d_img_3, uint8_t* d_img_out) {
  size_t i = blockIdx.x;
  size_t j = blockIdx.y * 512 + threadIdx.x;

  if (i < height && j < width) {
    size_t index_single = i * width + j;
    size_t index_1 = i * 3 * width + j;
    size_t index_2 = i * 3 * width + j + width;
    size_t index_3 = i * 3 * width + j + 2 * width;
    d_img_out[3 * index_1] = d_img_1[3 * index_single];
    d_img_out[3 * index_1 + 1] = d_img_1[3 * index_single + 1];
    d_img_out[3 * index_1 + 2] = d_img_1[3 * index_single + 2];
    d_img_out[3 * index_2] = d_img_2[3 * index_single];
    d_img_out[3 * index_2 + 1] = d_img_2[3 * index_single + 1];
    d_img_out[3 * index_2 + 2] = d_img_2[3 * index_single + 2];
    d_img_out[3 * index_3] = d_img_3[3 * index_single];
    d_img_out[3 * index_3 + 1] = d_img_3[3 * index_single + 1];
    d_img_out[3 * index_3 + 2] = d_img_3[3 * index_single + 2];
  }
}

__device__ void rgb2YUVPixel(uint8_t r, uint8_t g, uint8_t b, uint8_t& y, uint8_t& u, uint8_t& v) {
  float Y = (0.257 * (float)r) + (0.504 * (float)g) + (0.098 * (float)b) + 16;
  float V = (0.439 * (float)r) - (0.368 * (float)g) - (0.071 * (float)b) + 128;
  float U = -(0.148 * (float)r) - (0.291 * (float)g) + (0.439 * (float)b) + 128;
  y = (uint8_t)(clamp(Y, 0, 255));
  u = (uint8_t)(clamp(U, 0, 255));
  v = (uint8_t)(clamp(V, 0, 255));
}

// Thread per block: 1D - 512
// Blocks per grid: 2D - height / 2, (width / 2 + 511) / 512
__global__ void rgb2YUV4Pixel(size_t width, size_t height, uint8_t* d_img, uint8_t* d_img_out) {
  size_t i = blockIdx.x;
  size_t j = blockIdx.y * 512 + threadIdx.x;

  if (i < height / 2 && j < width / 2) {
    // Pixel 1
    size_t i_1 = 2 * i;
    size_t j_1 = 2 * j;
    size_t index_1 = i_1 * width + j_1;
    uint8_t y_1, u_1, v_1;
    rgb2YUVPixel(d_img[3 * index_1], d_img[3 * index_1 + 1], d_img[3 * index_1 + 2], y_1, u_1, v_1);
    d_img_out[index_1] = y_1;

    // Pixel 2
    size_t i_2 = 2 * i;
    size_t j_2 = 2 * j + 1;
    size_t index_2 = i_2 * width + j_2;
    uint8_t y_2, u_2, v_2;
    rgb2YUVPixel(d_img[3 * index_2], d_img[3 * index_2 + 1], d_img[3 * index_2 + 2], y_2, u_2, v_2);
    d_img_out[index_2] = y_2;

    // Pixel 3
    size_t i_3 = 2 * i + 1;
    size_t j_3 = 2 * j;
    size_t index_3 = i_3 * width + j_3;
    uint8_t y_3, u_3, v_3;
    rgb2YUVPixel(d_img[3 * index_3], d_img[3 * index_3 + 1], d_img[3 * index_3 + 2], y_3, u_3, v_3);
    d_img_out[index_3] = y_3;

    // Pixel 4
    size_t i_4 = 2 * i + 1;
    size_t j_4 = 2 * j + 1;
    size_t index_4 = i_4 * width + j_4;
    uint8_t y_4, u_4, v_4;
    rgb2YUVPixel(d_img[3 * index_4], d_img[3 * index_4 + 1], d_img[3 * index_4 + 2], y_4, u_4, v_4);
    d_img_out[index_4] = y_4;
    
    d_img_out[width * height + i * width / 2 + j] = u_1 / 4 + u_2 / 4 + u_3 / 4 + u_4 / 4;
    d_img_out[width * height + width * height / 4 + i * width / 2 + j] = v_1 / 4 + v_2 / 4 + v_3 / 4 + v_4 / 4;
  }
}

// Thread per block: 1D - 512
// Blocks per grid: 2D - height, (width / 2 + 511) / 512
__global__ void yuyv2YUV2Pixel(size_t width, size_t height, uint8_t* d_img, uint8_t* d_img_out) {
    size_t i = blockIdx.x;
    size_t j = blockIdx.y * 512 + threadIdx.x;
  
    if (i < height && j < width / 2) {
      d_img_out[i * width + 2 * j] = d_img[2 * width * i + 4 * j];
      d_img_out[i * width + 2 * j + 1] = d_img[2 * width * i + 4 * j + 2];
      d_img_out[width * height + i / 2 * width / 2 + j] = d_img[2 * width * i + 4 * j + 1];
      d_img_out[width * height + width * height / 4 + i / 2 * width / 2 + j] = d_img[2 * width * i + 4 * j + 3];
    }
  }

// Thread per block: 1D - 512
// Blocks per grid: 2D - height / 2, (width / 2 + 511) / 512
__global__ void shuffleYUVPixel(size_t width, size_t height, uint8_t* d_img, uint8_t* d_img_out) {
  size_t i = blockIdx.x;
  size_t j = blockIdx.y * 512 + threadIdx.x;

  if (i < height / 2 && j < width / 2) {
    uint8_t U = d_img[width * height + i * width / 2 + j];
    uint8_t V = d_img[width * height * 5 / 4 + i * width / 2 + j];
    if (j % 2 == 0) {
      d_img_out[width * height + i * width / 4 + j / 2] = U;
      d_img_out[width * height + height / 4 * width / 2 + i * width / 4 + j / 2] = V;
    } else {
      d_img_out[width * height * 5 / 4 + i * width / 4 + j / 2] = U;
      d_img_out[width * height * 5 / 4 + height / 4 * width / 2 + i * width / 4 + j / 2] = V;
    }
  }
}

// Thread per block: 1D - 512
// Blocks per grid: 2D - height, (width / 2 + 511) / 512
__global__ void copyYUYVPixelOffset(size_t width, size_t height, size_t width_dst, size_t offset, uint8_t* d_img, uint8_t* d_img_out) {
  size_t i = blockIdx.x;
  size_t j = blockIdx.y * 512 + threadIdx.x;

  if (i < height && j < width / 2) {
    d_img_out[i * width_dst * 2 + offset * 2 + 4 * j] = d_img[2 * width * i + 4 * j];
    d_img_out[i * width_dst * 2 + offset * 2 + 4 * j + 1] = d_img[2 * width * i + 4 * j + 1];
    d_img_out[i * width_dst * 2 + offset * 2 + 4 * j + 2] = d_img[2 * width * i + 4 * j + 2];
    d_img_out[i * width_dst * 2 + offset * 2 + 4 * j + 3] = d_img[2 * width * i + 4 * j + 3];
  }
}

}

uint8_t* allocateImage(size_t width, size_t height) {
  uint8_t* d_res = nullptr;
  hipMalloc(&d_res, width * height * 3);
  return d_res;  
}

uint8_t* allocateImageYUV(size_t width, size_t height) {
  uint8_t* d_res = nullptr;
  hipMalloc(&d_res, width * height * 3 / 2);
  return d_res;  
}

uint8_t* allocateImageYUYV(size_t width, size_t height) {
  uint8_t* d_res = nullptr;
  hipMalloc(&d_res, width * height * 2);
  return d_res; 
}

uint8_t* uploadImage(size_t width, size_t height, uint8_t* img) {
  uint8_t* d_res = allocateImage(width, height);
  CHECK(d_res);
  hipMemcpy(d_res, img, width * height * 3, hipMemcpyHostToDevice);
  return d_res;  
}

void uploadImage(size_t width, size_t height, uint8_t* img, uint8_t* d_img) {
  CHECK(img);
  CHECK(d_img);
  hipMemcpy(d_img, img, width * height * 3, hipMemcpyHostToDevice);
}

void downloadImage(size_t width, size_t height, uint8_t* d_img, uint8_t* img) {
  hipMemcpy(img, d_img, width * height * 3, hipMemcpyDeviceToHost);
}

void downloadImageYUV(size_t width, size_t height, uint8_t* d_img, uint8_t* img) {
  hipMemcpy(img, d_img, width * height * 3 / 2, hipMemcpyDeviceToHost);
}

void copyImage(uint8_t* d_dst, uint8_t* d_src, size_t size) {
  hipMemcpy(d_dst, d_src, size, hipMemcpyDeviceToDevice);
}

void freeImage(uint8_t* d_img) {
  hipFree(d_img);
}

void combineThreeImages(size_t width, size_t height, uint8_t* d_img_1, uint8_t* d_img_2, uint8_t* d_img_3, uint8_t* d_img_out) {
  CHECK(d_img_1);
  CHECK(d_img_2);
  CHECK(d_img_3);
  CHECK(d_img_out);
  dim3 gridDims(height, (width + 511) / 512);
  copyPixel3<<<gridDims, 512>>>(width, height, d_img_1, d_img_2, d_img_3, d_img_out);
}

void rgb2YUV(size_t width, size_t height, uint8_t* d_img, uint8_t* d_img_out) {
  CHECK(d_img);
  CHECK(d_img_out);
  dim3 gridDims(height / 2, (width / 2 + 511) / 512);
  rgb2YUV4Pixel<<<gridDims, 512>>>(width, height, d_img, d_img_out);
}

void yuyv2YUV(size_t width, size_t height, uint8_t* d_img, uint8_t* d_img_out) {
  CHECK(d_img);
  CHECK(d_img_out);
  dim3 gridDims(height, (width / 2 + 511) / 512);
  yuyv2YUV2Pixel<<<gridDims, 512>>>(width, height, d_img, d_img_out);
}

void shuffleYUV(size_t width, size_t height, uint8_t* d_img, uint8_t* d_img_out) {
  CHECK(d_img);
  CHECK(d_img_out);

  // Copy the lumination first
  copyImage(d_img_out, d_img, width * height);

  dim3 gridDims(height / 2, (width / 2 + 511) / 512);
  shuffleYUVPixel<<<gridDims, 512>>>(width, height, d_img, d_img_out);
}

void copyYUYVWithOffset(uint8_t* d_dst, size_t width_dst, uint8_t* d_src, size_t width, size_t height, size_t offset) {
  CHECK(d_dst);
  CHECK(d_src);

  dim3 gridDims(height, (width / 2 + 511) / 512);
  copyYUYVPixelOffset<<<gridDims, 512>>>(width, height, width_dst, offset, d_src, d_dst);
}

}
