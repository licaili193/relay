#include "hip/hip_runtime.h"
#include "image_processing.h"

#include "camera_grabber.h"

void CameraGrabber::worker() {
  CHECK(cuda_ctx_);
  CHECK_EQ(hipSuccess, hipCtxSetCurrent(*cuda_ctx_));
  hipSetDevice(0);

  // TODO: Has to be initialized in the worker?
  // 1.  Open the device
  int fd; // A file descriptor to the video device
  fd = open(camera_name_.c_str(), O_RDWR);
  if(fd < 0){
      LOG(FATAL) << "Failed to open device, OPEN";
  }


  // 2. Ask the device if it can capture frames
  v4l2_capability capability;
  if(ioctl(fd, VIDIOC_QUERYCAP, &capability) < 0){
      // something went wrong... exit
      LOG(FATAL) << "Failed to get device capabilities, VIDIOC_QUERYCAP";
  }
  
  // 3. Set Image format
  v4l2_format imageFormat;
  imageFormat.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
  imageFormat.fmt.pix.width = width_;
  imageFormat.fmt.pix.height = height_;
  imageFormat.fmt.pix.pixelformat = V4L2_PIX_FMT_YUYV;
  imageFormat.fmt.pix.field = V4L2_FIELD_INTERLACED;
  // tell the device you are using this format
  if(ioctl(fd, VIDIOC_S_FMT, &imageFormat) < 0){
      LOG(FATAL) << "Device could not set format, VIDIOC_S_FMT";
  }

  // 4. Request Buffers from the device
  v4l2_requestbuffers requestBuffer = {0};
  requestBuffer.count = 1; // one request buffer
  requestBuffer.type = V4L2_BUF_TYPE_VIDEO_CAPTURE; // request a buffer which we can use for capturing frames
  requestBuffer.memory = V4L2_MEMORY_MMAP;

  if(ioctl(fd, VIDIOC_REQBUFS, &requestBuffer) < 0){
      LOG(FATAL) << "Could not request buffer from device, VIDIOC_REQBUFS";
  }
  
  // 5. Query the buffer to get raw data ie. ask for the you requested buffer
  // and allocate memory for it
  v4l2_buffer queryBuffer = {0};
  queryBuffer.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
  queryBuffer.memory = V4L2_MEMORY_MMAP;
  queryBuffer.index = 0;
  if(ioctl(fd, VIDIOC_QUERYBUF, &queryBuffer) < 0){
      LOG(FATAL) << "Device did not return the buffer information, VIDIOC_QUERYBUF";
  }
  // use a pointer to point to the newly created buffer
  // mmap() will map the memory address of the device to
  // an address in memory
  char* buffer = (char*)mmap(NULL, queryBuffer.length, PROT_READ | PROT_WRITE, MAP_SHARED,
                      fd, queryBuffer.m.offset);
  memset(buffer, 0, queryBuffer.length);


  // 6. Get a frame
  // Create a new buffer type so the device knows which buffer we are talking about
  v4l2_buffer bufferinfo;
  memset(&bufferinfo, 0, sizeof(bufferinfo));
  bufferinfo.type = V4L2_BUF_TYPE_VIDEO_CAPTURE;
  bufferinfo.memory = V4L2_MEMORY_MMAP;
  bufferinfo.index = 0;

  // Activate streaming
  int type = bufferinfo.type;
  if(ioctl(fd, VIDIOC_STREAMON, &type) < 0){
      LOG(FATAL) << "Could not start streaming, VIDIOC_STREAMON";
  }

  running_.store(true);
  while (running_.load()) {
      // Queue the buffer
      if(ioctl(fd, VIDIOC_QBUF, &bufferinfo) < 0){
          LOG(ERROR) << "Could not queue buffer, VIDIOC_QBUF";
          continue;
      }

      // Dequeue the buffer
      if(ioctl(fd, VIDIOC_DQBUF, &bufferinfo) < 0){
          LOG(ERROR) << "Could not dequeue the buffer, VIDIOC_DQBUF";
          continue;
      }
      // Frames get written after dequeuing the buffer

      memcpy(out_buffer_, buffer, std::min(out_buffer_size_, (size_t)bufferinfo.bytesused));
      hipMemcpy(d_img_yuyv_, out_buffer_, out_buffer_size_, hipMemcpyHostToDevice);
      new_image_.store(true);
  }

  // end streaming
  if(ioctl(fd, VIDIOC_STREAMOFF, &type) < 0){
    LOG(ERROR) << "Could not end streaming, VIDIOC_STREAMOFF";
  }

  close(fd);
}
